#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename scalar_t>
__global__ void rational_fwd_cuda_kernel_1dgroup(
    const scalar_t* __restrict__ x, 
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b, 
    scalar_t* __restrict__ result, 
    int B, int L, int D, int group, 
    int x_size, int D_per_group) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= x_size) return;  // Prevent out-of-bounds memory access

    // Calculate the index within the dimension D
    int d_index = idx % D;
    // Calculate the group index based on the position within dimension D
    int g_index = d_index / D_per_group;

    // Calculate specific indices for a and b based on group
    int a_idx = g_index * 6;
    int b_idx = g_index * 4;

    // Load coefficients into registers
    scalar_t s_a[6], s_b[4];
    for (int i = 0; i < 6; ++i) {
        s_a[i] = a[a_idx + i];
    }
    for (int i = 0; i < 4; ++i) {
        s_b[i] = abs(b[b_idx + i]);  // Store absolute values directly if needed
    }

    // Obtain the input value from the tensor
    scalar_t xp1 = x[idx];
    scalar_t abs_xp1 = abs(xp1);

    // Compute the polynomial for P using Horner's method
    scalar_t P = s_a[5];
    for (int i = 4; i >= 0; --i) {
        P = fmaf(P, xp1, s_a[i]);
    }
    
    // Compute the polynomial for Q using Horner's method
    scalar_t Q = s_b[3];
    for (int i = 2; i >= 0; --i) {
        Q = fmaf(Q, abs_xp1, s_b[i]);
    }
    Q = fmaf(Q, abs_xp1, 1.0);

    // Write the result of P / Q
    result[idx] = P / Q;
}

torch::Tensor rational_fwd_cuda_1dgroup(
    torch::Tensor x, 
    torch::Tensor n, 
    torch::Tensor d,
    int group
    ){
    auto result = at::empty_like(x);
    const int x_size = x.numel();
    int B = x.size(0);
    int L = x.size(1);
    int D = x.size(2);

    int threads_per_block = 256;  // Adjust as needed based on device capabilities
    int num_blocks = (x_size + threads_per_block - 1) / threads_per_block;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "rational_fwd_cuda_1dgroup", ([&] {
    rational_fwd_cuda_kernel_1dgroup<scalar_t>
        <<<num_blocks, threads_per_block>>>(
            x.data_ptr<scalar_t>(),
            n.data_ptr<scalar_t>(),
            d.data_ptr<scalar_t>(),
            result.data_ptr<scalar_t>(),
            B, L, D, group, x_size, D / group);
        }));

    return result;
}

//P(X) = a_0 + a_1*X + a_2*X^2 ...
//Q(X) = 1 + |b_0||X| + |b_1||X|^2 + |b_2||X|^3
//R(X) = a_1 + 2*a_2*X + 3*a_3*X ...
//S(X) = sign(X) * ( |b_0| + 2|b_1||X| + 3|b_2||X|^2 ...)
//dF/dx = (-P(X)/Q(X)^2)*S(X) + R(X)/Q(X)
//dF/da_i = x^i/Q(X), i \in {0,5}
//dF/db_i = (-P(X)/Q(X)^2) * sign(b_i) * |X^{i+1}| , i \in {0,4}


template <typename scalar_t>
__global__ void rational_bwd_cuda_kernel_1dgroup(
    const scalar_t* __restrict__ grad_output,
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ d_x,
    double* __restrict__ d_a,
    double* __restrict__ d_b,
    int B, int L, int D, int group, 
    int x_size, int D_per_group) {
    
    __shared__ double sda[6 * group];
    __shared__ double sdb[4 * group];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= x_size) return;  // Prevent out-of-bounds memory access

    // Calculate the index within the dimension D
    int d_index = idx % D;
    // Calculate the group index based on the position within dimension D
    int g_index = d_index / D_per_group;

    // Calculate specific indices for a and b based on group
    int a_idx = g_index * 6;
    int b_idx = g_index * 4;

    // Load coefficients into registers
    scalar_t shared_a[6], shared_b_abs[4];
    for (int i = 0; i < 6; ++i) {
        shared_a[i] = a[a_idx + i];
    }
    for (int i = 0; i < 4; ++i) {
        shared_b_abs[i] = abs(b[b_idx + i]);  // Store absolute values directly if needed
    }

    double local_da[6] = {0}; // Local accumulation arrays
    double local_db[4] = {0};
    
    scalar_t xp = x[idx];
    scalar_t axp = abs(xp);
    // Compute powers of xp
    scalar_t xp_powers[5];
    xp_powers[0] = xp;
    xp_powers[1] = xp * xp_powers[0]; // xp^2
    xp_powers[2] = xp * xp_powers[1]; // xp^3
    xp_powers[3] = xp * xp_powers[2]; // xp^4
    xp_powers[4] = xp * xp_powers[3]; // xp^5

    // Compute powers of axp
    scalar_t axp_powers[4];
    axp_powers[0] = axp;
    axp_powers[1] = abs(xp_powers[1]); // axp^2
    axp_powers[2] = abs(xp_powers[2]); // axp^3
    axp_powers[3] = abs(xp_powers[3]); // axp^4

    // Compute absolute values once

    scalar_t P = shared_a[0] 
    + shared_a[1] * xp_powers[0] 
    + shared_a[2] * xp_powers[1] 
    + shared_a[3] * xp_powers[2] 
    + shared_a[4] * xp_powers[3] 
    + shared_a[5] * xp_powers[4];

    scalar_t Q = 1.0 
    + shared_b_abs[0] * axp_powers[0] 
    + shared_b_abs[1] * axp_powers[1] 
    + shared_b_abs[2] * axp_powers[2] 
    + shared_b_abs[3] * axp_powers[3];
    
    scalar_t Q_inv = 1.0 / Q;
    scalar_t Q_inv2 = Q_inv * Q_inv;

    scalar_t grad_o = grad_output[idx];
    
    scalar_t R = shared_a[1] 
    + 2.0 * shared_a[2] * xp_powers[0] 
    + 3.0 * shared_a[3] * xp_powers[1] 
    + 4.0 * shared_a[4] * xp_powers[2] 
    + 5.0 * shared_a[5] * xp_powers[3];

    scalar_t S = copysign(1.0, xp) * (shared_b_abs[0] 
    + 2.0 * shared_b_abs[1] * axp_powers[0] 
    + 3.0 * shared_b_abs[2] * axp_powers[1] 
    + 4.0 * shared_b_abs[3] * axp_powers[2]);

    scalar_t d_i_x = (R * Q_inv + S * (-P * Q_inv2)) * grad_o;
    d_x[idx] = d_i_x;

    // Precompute common factors outside the loops
    scalar_t common_factor_da = Q_inv * grad_o;
    scalar_t common_factor_db = (-P * Q_inv2) * grad_o;

    // Loop for computing d_a contributions
    for (int i = 0; i < 6; ++i) {
        local_da[i] += xp_powers[i] * common_factor_da;
    }

    // Loop for computing d_b contributions
    for (int i = 0; i < 4; ++i) {
        local_db[i] += copysign(1.0, b[i]) * axp_powers[i] * common_factor_db;
    }

    // Reduce local arrays to shared memory
    for (int i = 0; i < 6; ++i) {
        atomicAdd(&sda[a_idx + i], local_da[i]);
    }
    for (int i = 0; i < 4; ++i) {
        atomicAdd(&sdb[b_idx + i], local_db[i]);
    }

    __syncthreads();

    // Only one thread writes back to global memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < 6 * group; ++i) {
            atomicAdd(&d_a[i], sda[i]);
        }
        for (int i = 0; i < 4 * group; ++i) {
            atomicAdd(&d_b[i], sdb[i]);
        }
    }
}

std::vector<torch::Tensor> rational_bwd_cuda_1dgroup(torch::Tensor grad_output, torch::Tensor x, torch::Tensor n, torch::Tensor d, int group) {
    const int x_size = x.numel();
    auto d_x = at::empty_like(x);
    auto d_n = at::zeros_like(n).toType(at::kDouble);
    auto d_d = at::zeros_like(d).toType(at::kDouble);

    int B = x.size(0);
    int L = x.size(1);
    int D = x.size(2);

    int blockSize = 256;  // You might want to experiment with this value
    int numBlocks = (x_size + blockSize - 1) / blockSize;

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "rational_bwd_cuda_1dgroup", ([&] {
    rational_bwd_cuda_kernel_1dgroup<scalar_t>
        <<<numBlocks, blockSize>>>(
            grad_output.data_ptr<scalar_t>(),
            x.data_ptr<scalar_t>(),
            n.data_ptr<scalar_t>(),
            d.data_ptr<scalar_t>(),
            d_x.data_ptr<scalar_t>(),
            d_n.data_ptr<double>(),
            d_d.data_ptr<double>(),
            B, L, D, group, x_size, D / group);
    }));

    return {d_x, d_n.toType(at::kFloat), d_d.toType(at::kFloat)};
}