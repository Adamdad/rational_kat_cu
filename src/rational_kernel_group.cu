#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename scalar_t>
__global__ void rational_fwd_cuda_kernel_1dgroup(
    const scalar_t* __restrict__ x, 
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b, 
    scalar_t* __restrict__ result, 
    int B, int L, int D, int group, 
    int x_size, int D_per_group) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= x_size) return;  // Prevent out-of-bounds memory access

    // Calculate the index within the dimension D
    int d_index = idx % D;
    // Calculate the group index based on the position within dimension D
    int g_index = floor(d_index / D_per_group);

    // Calculate specific indices for a and b based on group
    int a_idx = g_index * 6;
    int b_idx = g_index * 4;

    // Load coefficients into registers
    scalar_t s_a[6], s_b[4];
    for (int i = 0; i < 6; ++i) {
        s_a[i] = a[a_idx + i];
    }
    for (int i = 0; i < 4; ++i) {
        s_b[i] = abs(b[b_idx + i]);  // Store absolute values directly if needed
    }

    // Obtain the input value from the tensor
    scalar_t xp1 = x[idx];
    scalar_t abs_xp1 = abs(xp1);

    // Compute the polynomial for P using Horner's method
    scalar_t P = s_a[5];
    for (int i = 4; i >= 0; --i) {
        P = fmaf(P, xp1, s_a[i]);
    }
    
    // Compute the polynomial for Q using Horner's method
    scalar_t Q = s_b[3];
    for (int i = 2; i >= 0; --i) {
        Q = fmaf(Q, abs_xp1, s_b[i]);
    }
    Q = fmaf(Q, abs_xp1, 1.0);

    // Write the result of P / Q
    result[idx] = P / Q;
}

torch::Tensor rational_fwd_cuda_1dgroup(
    torch::Tensor x, 
    torch::Tensor n, 
    torch::Tensor d,
    int group
    ){
    auto result = at::empty_like(x);
    const int x_size = x.numel();
    int B = x.size(0);
    int L = x.size(1);
    int D = x.size(2);

    int threads_per_block = 256;  // Adjust as needed based on device capabilities
    int num_blocks = (x_size + threads_per_block - 1) / threads_per_block;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "rational_fwd_cuda_1dgroup", ([&] {
    rational_fwd_cuda_kernel_1dgroup<scalar_t>
        <<<num_blocks, threads_per_block>>>(
            x.data_ptr<scalar_t>(),
            n.data_ptr<scalar_t>(),
            d.data_ptr<scalar_t>(),
            result.data_ptr<scalar_t>(),
            B, L, D, group, x_size, D / group);
        }));

    return result;
}

//P(X) = a_0 + a_1*X + a_2*X^2 ...
//Q(X) = 1 + |b_0||X| + |b_1||X|^2 + |b_2||X|^3
//R(X) = a_1 + 2*a_2*X + 3*a_3*X ...
//S(X) = sign(X) * ( |b_0| + 2|b_1||X| + 3|b_2||X|^2 ...)
//dF/dx = (-P(X)/Q(X)^2)*S(X) + R(X)/Q(X)
//dF/da_i = x^i/Q(X), i \in {0,5}
//dF/db_i = (-P(X)/Q(X)^2) * sign(b_i) * |X^{i+1}| , i \in {0,4}


template <typename scalar_t>
__global__ void rational_bwd_cuda_kernel_1dgroup(
    const scalar_t* __restrict__ grad_output,
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ d_x,
    float* __restrict__ d_a,
    float* __restrict__ d_b,
    int B, int L, int D, int group, 
    int x_size, 
    const int n_size, 
    const int d_size,
    int D_per_group) {
    
    // Shared memory for accumulation
    // group < 32, so we can use 192 and 128 shared memory
    __shared__ float sda[192];
    __shared__ float sdb[128];
    // initialize shared memory to zero
    if ( threadIdx.x == 0) {
        for (int i = 0; i < 192; ++i) {
            sda[i] = 0;
        }
        for (int i = 0; i < 128; ++i) {
            sdb[i] = 0;
        }
    }

    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= x_size) return;  // Prevent out-of-bounds memory access

    // Calculate the index within the dimension D
    int d_index = idx % D;
    // Calculate the group index based on the position within dimension D
    int g_index = floor(d_index / D_per_group);

    // Calculate specific indices for a and b based on group
    int a_idx = g_index * 6;
    int b_idx = g_index * 4;

    // Load coefficients into registers
    scalar_t shared_a[6], shared_b_abs[4], shared_b[4];;
    for (int i = 0; i < 6; ++i) {
        shared_a[i] = a[a_idx + i];
    }
    for (int i = 0; i < 4; ++i) {
        shared_b_abs[i] = abs(b[b_idx + i]);  // Store absolute values directly if needed
        shared_b[i] = b[b_idx + i];
    }

    scalar_t local_da[6] = {0}; // Local accumulation arrays
    scalar_t local_db[4] = {0};
    
    scalar_t xp = x[idx];
    scalar_t axp = abs(xp);
    // Compute powers of xp
    scalar_t xp_powers[5];
    xp_powers[0] = xp;
    xp_powers[1] = xp * xp_powers[0]; // xp^2
    xp_powers[2] = xp * xp_powers[1]; // xp^3
    xp_powers[3] = xp * xp_powers[2]; // xp^4
    xp_powers[4] = xp * xp_powers[3]; // xp^5

    // Compute powers of axp
    scalar_t axp_powers[4];
    axp_powers[0] = axp;
    axp_powers[1] = axp * axp_powers[0]; // axp^2
    axp_powers[2] = axp * axp_powers[1]; // axp^3
    axp_powers[3] = axp * axp_powers[2]; // axp^4

    // Compute absolute values once

    scalar_t P = shared_a[0] 
    + shared_a[1] * xp_powers[0] 
    + shared_a[2] * xp_powers[1] 
    + shared_a[3] * xp_powers[2] 
    + shared_a[4] * xp_powers[3] 
    + shared_a[5] * xp_powers[4];

    scalar_t Q = scalar_t(1.0)
    + shared_b_abs[0] * axp_powers[0] 
    + shared_b_abs[1] * axp_powers[1] 
    + shared_b_abs[2] * axp_powers[2] 
    + shared_b_abs[3] * axp_powers[3];


    scalar_t R = shared_a[1] 
    + scalar_t(2.0) * shared_a[2] * xp_powers[0] 
    + scalar_t(3.0) * shared_a[3] * xp_powers[1] 
    + scalar_t(4.0) * shared_a[4] * xp_powers[2] 
    + scalar_t(5.0) * shared_a[5] * xp_powers[3];

    scalar_t S = copysign(scalar_t(1.0), xp) * (shared_b_abs[0] 
    + scalar_t(2.0) * shared_b_abs[1] * axp_powers[0] 
    + scalar_t(3.0) * shared_b_abs[2] * axp_powers[1] 
    + scalar_t(4.0) * shared_b_abs[3] * axp_powers[2]);
    

    scalar_t grad_o = grad_output[idx];
    
    scalar_t mpq2 = -P/(Q*Q);

    scalar_t d_i_x = (R / Q + S * mpq2) * grad_o;
    d_x[idx] = d_i_x;

    // Loop for computing d_a contributions
    local_da[0] = scalar_t(1.0) / Q * grad_o;
    for (int i = 1; i < 6; ++i) {
        local_da[i] = (xp_powers[i-1] / Q) * grad_o;
    }

    // Loop for computing d_b contributions
    for (int i = 0; i < 4; ++i) {
        local_db[i] = mpq2 * copysign(scalar_t(1.0), shared_b[i]) * axp_powers[i] * grad_o;
    }

    // Reduce local arrays to shared memory
    for (int i = 0; i < 6; ++i) {
        atomicAdd(&sda[a_idx + i], local_da[i]);
    }
    for (int i = 0; i < 4; ++i) {
        atomicAdd(&sdb[b_idx + i], local_db[i]);
    }

    __syncthreads();

    // Only one thread writes back to global memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < n_size; ++i) {
            atomicAdd(&d_a[i], sda[i]);
        }
        for (int i = 0; i < d_size; ++i) {
            atomicAdd(&d_b[i], sdb[i]);
        }
    }
}

std::vector<torch::Tensor> rational_bwd_cuda_1dgroup(torch::Tensor grad_output, torch::Tensor x, torch::Tensor n, torch::Tensor d, int group) {
    const int x_size = x.numel();
    const int n_size = n.numel();
    const int d_size = d.numel();

    auto d_x = at::empty_like(x);
    auto d_n = at::zeros_like(n).toType(at::kFloat);
    auto d_d = at::zeros_like(d).toType(at::kFloat);

    int B = x.size(0);
    int L = x.size(1);
    int D = x.size(2);

    int blockSize = 256;  // You might want to experiment with this value
    int numBlocks = (x_size + blockSize - 1) / blockSize;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "rational_bwd_cuda_1dgroup", ([&] {
    rational_bwd_cuda_kernel_1dgroup<scalar_t>
        <<<numBlocks, blockSize>>>(
            grad_output.data_ptr<scalar_t>(),
            x.data_ptr<scalar_t>(),
            n.data_ptr<scalar_t>(),
            d.data_ptr<scalar_t>(),
            d_x.data_ptr<scalar_t>(),
            d_n.data_ptr<float>(),
            d_d.data_ptr<float>(),
            B, L, D, group, x_size, n_size, d_size, D / group);
    }));

    return {d_x, d_n, d_d};
}